#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

//compute a*b mod p safely
__device__ unsigned int modprodC(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  } // end while
  return ab;
} //end modprodC function

//compute a^b mod p safely
__device__ unsigned int modExpC(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modprodC(aExpb, z, p);
    z = modprodC(z, z, p);
    b /= 2;
  } // end while
  return aExpb;
} // end modprodC function

  __global__ void findSecretKey(unsigned int p, unsigned int g, unsigned int h, unsigned int *d_a) {
  
  int threadId = threadIdx.x;
  int blockId = blockIdx.x;
  int Nblock = blockDim.x;
 
  int id = threadId+blockId*Nblock;

  if (id < (p-1)) {
	if(modExpC(g, id, p) == h) {
		*d_a = id;
	} // end inner if
  } // end outer if

 } // end findKey


int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from your completed decrypt.c main function. */
  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  
  FILE* pk = fopen("bonus_public_key.txt","r");
  FILE* msg = fopen("bonus_message.txt", "r");
  fscanf(pk, "%u\n%u\n%u\n%u\n", &n, &p, &g, &h);
  fclose(pk);
  fscanf(msg, "%u\n", &Nints);

  unsigned int *mhat = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));

   for (int i = 0; i < Nints; i++) {
    fscanf(msg, "%u %u\n", &mhat[i], &a[i]);
  } // end for

  fclose(msg);
 // } // end main

  /* Q3 After finding the secret key, decrypt the message */

  unsigned int found;
  unsigned int *d_a;

  hipMalloc(&d_a, sizeof(unsigned int));

    if (x == 0 || modExp(g, x, p) != h) {
	printf("Finding the secret key ... \n");
  } // end if

  double startTime = clock();

  unsigned int Nthreads = 32;
  unsigned int Nblocks = (p-1)/Nthreads;

  findSecretKey <<<Nblocks, Nthreads>>>(g, h, p, d_a);
  hipDeviceSynchronize();

  hipMemcpy(&found, d_a, sizeof(unsigned int), hipMemcpyDeviceToHost);

  double endTime = clock();
  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
  double work = (double) p;
  double throughput = work/totalTime;

  printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);

  printf("Secret key found! x = %u \n", found);

  int Nchars = ((n-1)/8)*Nints;
  unsigned char *decrypted = (unsigned char *) malloc(1024*sizeof(unsigned char));

  ElGamalDecrypt(mhat, a, Nints, p, found);
  convertZToString(mhat, Nints, decrypted, Nchars);
  printf("Decrypted message: %s\n", decrypted);
    
  hipFree(d_a);
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
  
  return 0;
}
